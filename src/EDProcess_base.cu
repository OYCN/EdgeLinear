#include "EDProcess_par.h"
#include "Timer.h"
#include <assert.h>

/*
*Summary: 构造函数
*Parameters: 
*    _rows: 待处理图像行数
*    _cols: 待处理图像列数
*    _anchor_th: 锚点阈值
*    _k: 锚点稀疏程度
*Return: 无
*/
Main::Main(int _rows, int _cols, int _anchor_th, int _k)
:rows(_rows), cols(_cols), anchor_th(_anchor_th), k(_k)
{
	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	if(count!=1) {printf( "zero or multiple gpu\n"); exit( EXIT_FAILURE );}
	hipSetDevice(0);
	// hipFree(0);

	// dimGridOld_FULL = dim3(rows, cols);
	_InitED();
	_InitPD();
}

/*
*Summary: 析构函数
*Parameters: 无
*Return: 无
*/
Main::~Main()
{
	_FreeED();
	_FreePD();
}

/*
*Summary: 类所需内存的申请等初始化操作，一个实例仅需一次
*Parameters: 无
*Return: 无
*/
void Main::_InitED()
{
	dimBlock_ED = dim3(32,32);
	dimGrid_ED = dim3((cols+27)/28, (rows+27)/28);
	HANDLE_ERROR(hipMalloc(&gMapd, sizeof(uchar)*rows*cols));
	HANDLE_ERROR(hipMalloc(&blurd, sizeof(uchar)*rows*cols));
	HANDLE_ERROR(hipMalloc(&fMapd, sizeof(uchar)*rows*cols));
	HANDLE_ERROR(hipMemset(gMapd, 0, sizeof(uchar)*rows*cols));
	HANDLE_ERROR(hipMemset(fMapd, 0, sizeof(uchar)*rows*cols));
	gMaph = new uchar[rows*cols];
	fMaph = new uchar[rows*cols];
	eMaph = cv::Mat::zeros(rows, cols, CV_8UC1);
	edge_set = new POINT[rows*cols];
	edge_offset = new int[rows*cols+1];
	edge_smart = new POINT[rows*cols];
}
/*
*Summary: 释放内存
*Parameters: 无
*Return: 无
*/
void Main::_FreeED()
{
	hipFree(gMapd);
	hipFree(blurd);
	hipFree(fMapd);
	delete[] gMaph;
	delete[] fMaph;
	delete[] edge_set;
	delete[] edge_offset;
	delete[] edge_smart;
}

void Main::setTH(int value)
{
	anchor_th = value;
}

int Main::getTH()
{
	return anchor_th;
}

/*
*Summary: 对原图像进行预处理
*Parameters: 
*    src: 原图像
*Return: 无
*/
void Main::PerProcED(cv::Mat &src)
{
	cv::cvtColor(src, grayImg, CV_RGB2GRAY);
	cv::GaussianBlur(grayImg, blurImg, cv::Size(5, 5), 1, 0);
}

/*
*Summary: 对传入图像进行单次边缘提取
*Parameters:
*     src: 传入图像
*     edge_seg: 用于保存边缘信息
*     edge_seg_offset: 每个线段在edge_seg中的偏移量
*     edge_seg_len: edge_seg_offset的长度
*Return: 边缘信息的图像
*/
cv::Mat Main::Process(cv::Mat& src, POINT *&edge_seg, int *&edge_seg_offset, int &edge_seg_len)
{
	// 确保图像尺寸与内存相符
	assert(src.rows == rows);
	assert(src.cols == cols);

	memset(eMaph.data, 0, rows*cols*sizeof(uchar));
	// cv::imshow("eMaph", eMaph*255);

	PerProcED(src);

	// 本次数据拷贝
	HANDLE_ERROR(hipMemcpy(blurd, blurImg.data, sizeof(uchar)*rows*cols, hipMemcpyHostToDevice));

	// 核函数启动
	kernelC<<< dimGrid_ED, dimBlock_ED >>>(blurd, gMapd, fMapd, cols, rows, anchor_th, k);
	
	// 核函数同步
	HANDLE_ERROR(hipDeviceSynchronize());

	// 数据拷回主内存
	HANDLE_ERROR(hipMemcpy(gMaph, gMapd, sizeof(uchar)*rows*cols, hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(fMaph, fMapd, sizeof(uchar)*rows*cols, hipMemcpyDeviceToHost));

	// 锚点连接，边缘提取
	cv::Mat eMap = smartConnecting();

	// cv::imshow("eMap", eMap*255);

	edge_seg = edge_set;
	edge_seg_offset = edge_offset;
	edge_seg_len = edge_offset_len;

	return eMap;
}
