#include "hip/hip_runtime.h"
#include "EdgeDrawing.h"

#define LIDX(x, y) [(x) + (y)*lcols]
#define GIDX(x, y) [(x) + (y)*gcols]

__global__ void kernelC(uchar *blur, uchar * gMap, uchar *fMap, int cols, int rows, int ANCHOR_TH, int K);

EdgeDrawing::EdgeDrawing(int _rows, int _cols, float _th, int _k)
    :rows(_rows), cols(_cols), th(_th), k(_k)
{
	HANDLE_ERROR(hipMalloc(&gMapd, sizeof(uchar)*rows*cols));
	HANDLE_ERROR(hipMalloc(&srcd, sizeof(uchar)*rows*cols));
	HANDLE_ERROR(hipMalloc(&fMapd, sizeof(uchar)*rows*cols));
	HANDLE_ERROR(hipMemset(gMapd, 0, sizeof(uchar)*rows*cols));
	HANDLE_ERROR(hipMemset(fMapd, 0, sizeof(uchar)*rows*cols));
	gMaph = new uchar[rows*cols];
	fMaph = new uchar[rows*cols];
	eMaph = new uchar[rows*cols];
	EDoutput.edge_set = new POINT[rows*cols];
	EDoutput.edge_offset = new int[rows*cols+1];
	edge_smart = new POINT[rows*cols];
    EDoutput.eMap = eMaph;
}

EdgeDrawing::~EdgeDrawing()
{
    hipFree(gMapd);
	hipFree(srcd);
	hipFree(fMapd);
	delete[] gMaph;
	delete[] fMaph;
    delete[] eMaph;
	delete[] EDoutput.edge_set;
	delete[] EDoutput.edge_offset;
	delete[] edge_smart;
}

void EdgeDrawing::initLoop()
{
    memset(eMaph, 0, rows*cols*sizeof(uchar));
}

_EDoutput* EdgeDrawing::run(cv::Mat& _src)
{
	// GPU Block 划分
    const dim3 dimBlock(32,32);;
    // GPU Grid 划分
    const dim3 dimGrid((cols+27)/28, (rows+27)/28);

    initLoop();
    
    cv::cvtColor(_src, srch, CV_RGB2GRAY);
	cv::GaussianBlur(srch, srch, cv::Size(5, 5), 1, 0);
	HANDLE_ERROR(hipMemcpy(srcd, srch.data, sizeof(uchar)*rows*cols, hipMemcpyHostToDevice));
    kernelC<<< dimGrid, dimBlock >>>(srcd, gMapd, fMapd, cols, rows, th, k);
    // HANDLE_ERROR(hipDeviceSynchronize());
    HANDLE_ERROR(hipMemcpy(gMaph, gMapd, sizeof(uchar)*rows*cols, hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(fMaph, fMapd, sizeof(uchar)*rows*cols, hipMemcpyDeviceToHost));
    // cv::Mat fMap(rows ,cols, CV_8UC1, (unsigned char*)(fMaph));
	// cv::imshow("fMap", fMap);
    smartConnecting();

	return &EDoutput;
}

__global__ void kernelC(uchar *blur, uchar * gMap, uchar *fMap, int gcols, int grows, int ANCHOR_TH, int K)
{
    int gx = blockIdx.x*28 + threadIdx.x;
    int gy = blockIdx.y*28 + threadIdx.y;
    const int &lx = threadIdx.x;
    const int &ly = threadIdx.y;
    const int &lcols = blockDim.x;
    const int &lrows = blockDim.y;
    int dx = 0;
    int dy = 0;
    float val = 0;
    uchar dir = 0;
	uchar flag1 = 0;
	uchar flag2 = 0;
	int com = 0;
	uchar center = 0;
    uchar fmap = 0;
    // uchar &a = flag1;
    // uchar &b = flag2;
    // uchar &c = center;
    __shared__ volatile uchar sblur[32*32];
    __shared__ volatile uchar sgMap[32*32];

    if(gx<gcols && gy<grows)
        sblur LIDX(lx, ly) = blur GIDX(gx, gy);
    __syncthreads();
    // 梯度计算
	if(lx!=0 && ly!=0 && lx<(lcols-1) && ly<(lrows-1) && gx<(gcols-1) && gy<(grows-1))
    {
        dx = sblur LIDX(lx+1,ly-1);
        dx += 2 * sblur LIDX(lx+1,ly);
        dx += sblur LIDX(lx+1,ly+1);
        dx -= sblur LIDX(lx-1,ly-1);
        dx -= 2 * sblur LIDX(lx-1,ly);
        dx -= sblur LIDX(lx-1,ly+1);
        dx = abs(dx);

        dy = sblur LIDX(lx-1,ly-1);
        dy += 2 * sblur LIDX(lx,ly-1);
        dy += sblur LIDX(lx+1,ly-1);
        dy -= sblur LIDX(lx-1,ly+1);
        dy -= 2 * sblur LIDX(lx,ly+1);
        dy -= sblur LIDX(lx+1,ly+1);
        dy = abs(dy);

        val = 0.5f*dx + 0.5f*dy;
        if (val > 255) val = 255.0f;

        center = (uchar)(val);
        sgMap LIDX(lx,ly) = center;
        gMap GIDX(gx,gy) = center;

        // 1 -- vertical   0 -- horizonal
        dir = dx > dy;
        fmap |= (dir<<7)&0x80;
    }
	__syncthreads();
    // 锚点提取
	if((lx>1 || gx==1) && (ly>1 || gy==1) && (lx<(lcols-2) || gx==(gcols-2)) && (ly<(lrows-2) || gy==(grows-2)) && gx<(gcols-1) && gy<(grows-1))
	{
		// h
		flag1 = !dir;
		com = center;
		com -= sgMap LIDX(lx, ly-1);
		flag1 &= com>=ANCHOR_TH;
		com = center;
		com -= sgMap LIDX(lx, ly+1);
		flag1 &= com>=ANCHOR_TH;
		// v
		flag2 = dir;
		com = center;
		com -= sgMap LIDX(lx-1,ly);
		flag2 &= com >= ANCHOR_TH;
		com = center;
		com -= sgMap LIDX(lx+1,ly);
		flag2 &= com >= ANCHOR_TH;
        fmap |= (((flag1 | flag2) && ((gx-1)%K)==0 && ((gy-1)%K)==0)<<6)&0x40;

        // 方向分析
        // 方案1 用于GPU计算结果
        // 	0	0	0	0	0	0	0	0
        // 					----- A方向（左/上）上/左、下/右
        // 							----- B方向（右/下）上/左、下/右

        // dir : 1 -- vertical   0 -- horizonal

        //      第一轮       第二轮
        //   h-0    v-1    h-0   v-1
        // a 左上 / 上左 | 右上 / 下左
        // b 左   / 上   | 右   / 下
        // c 左下 / 上右 | 右下 / 下右
        // a = sgMap LIDX(lx-1, ly-1);
        // b = sgMap LIDX(lx-1, ly) * !dir;
        // b += sgMap LIDX(lx, ly-1) * dir;
        // c = sgMap LIDX(lx-1, ly+1) * !dir;
        // c += sgMap LIDX(lx+1, ly-1) * dir;

        // fmap |= (a>b && a>c)

        fMap GIDX(gx,gy) = fmap;
	}

}