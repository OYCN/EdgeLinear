#include "hip/hip_runtime.h"
#include "EdgeDrawing.h"
// #include "Timer.h"

#define LIDX(x, y) [(x) + (y)*lcols]
#define GIDX(x, y) [(x) + (y)*gcols]

__global__ void kernelC(uchar *blur, uchar * gMap, uchar *fMap, int cols, int rows, int ANCHOR_TH, int K);

EdgeDrawing::EdgeDrawing(int _rows, int _cols, float _th, int _k)
    :rows(_rows), cols(_cols), th(_th), k(_k)
{
    HANDLE_ERROR(hipFree(0));
	HANDLE_ERROR(hipMalloc(&gMapd, sizeof(uchar)*rows*cols));
	HANDLE_ERROR(hipMalloc(&srcd, sizeof(uchar)*rows*cols));
	HANDLE_ERROR(hipMalloc(&fMapd, sizeof(uchar)*rows*cols));
	HANDLE_ERROR(hipMemset(gMapd, 0, sizeof(uchar)*rows*cols));
	HANDLE_ERROR(hipMemset(fMapd, 0, sizeof(uchar)*rows*cols));
	gMaph = new uchar[rows*cols];
	fMaph = new uchar[rows*cols];
	eMaph = new uchar[rows*cols];
	EDoutput.edge_set = new POINT[rows*cols];
	EDoutput.edge_offset = new int[rows*cols+1];
	edge_smart = new POINT[rows*cols];
    EDoutput.eMap = eMaph;
}

EdgeDrawing::~EdgeDrawing()
{
    hipFree(gMapd);
	hipFree(srcd);
	hipFree(fMapd);
	delete[] gMaph;
	delete[] fMaph;
    delete[] eMaph;
	delete[] EDoutput.edge_set;
	delete[] EDoutput.edge_offset;
	delete[] edge_smart;
}

void EdgeDrawing::initLoop()
{
    memset(eMaph, 0, rows*cols*sizeof(uchar));
}

_EDoutput* EdgeDrawing::run(cv::Mat& _src)
{
    // TDEF(gpu)
    // TDEF(init)
    // TDEF(h2d)
    // TDEF(kernel)
    // TDEF(d2h)
    // TDEF(cpu)
    // TSTART(gpu)
    // TSTART(init)
	// GPU Block 划分
    const dim3 dimBlock(32,32);;
    // GPU Grid 划分
    const dim3 dimGrid((cols+27)/28, (rows+27)/28);

    initLoop();
    
    cv::cvtColor(_src, srch, CV_RGB2GRAY);
	cv::GaussianBlur(srch, srch, cv::Size(5, 5), 1, 0);
    // TEND(init)
    // TSTART(h2d)
	HANDLE_ERROR(hipMemcpy(srcd, srch.data, sizeof(uchar)*rows*cols, hipMemcpyHostToDevice));
    // TEND(h2d)
    // TSTART(kernel)
    kernelC<<< dimGrid, dimBlock >>>(srcd, gMapd, fMapd, cols, rows, th, k);
    // HANDLE_ERROR(hipDeviceSynchronize());
    // TEND(kernel)
    // TSTART(d2h)
    HANDLE_ERROR(hipMemcpy(gMaph, gMapd, sizeof(uchar)*rows*cols, hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(fMaph, fMapd, sizeof(uchar)*rows*cols, hipMemcpyDeviceToHost));
    // TEND(d2h)
    // cv::Mat fMap(rows ,cols, CV_8UC1, (unsigned char*)(fMaph));
	// cv::imshow("fMap", fMap);
    // TEND(gpu)
    // TSTART(cpu)
    smartConnecting();
    // TEND(cpu)
    // TPRINTMS(gpu, "gpu:")
    // TPRINTMS(init, "\tinit:")
    // TPRINTMS(h2d, "\th2d:")
    // TPRINTMS(kernel, "\tkernel:")
    // TPRINTMS(d2h, "\td2h:")
    // TPRINTMS(cpu, "cpu:")

	return &EDoutput;
}

__global__ void kernelC(uchar *blur, uchar * gMap, uchar *fMap, int gcols, int grows, int ANCHOR_TH, int K)
{
    const int &lx = threadIdx.x;
    const int &ly = threadIdx.y;
    const int &lcols = blockDim.x;
    const int &lrows = blockDim.y;
    int gx = blockIdx.x*(lcols - 4) + threadIdx.x;
    int gy = blockIdx.y*(lrows - 4) + threadIdx.y;
    int dx = 0;
    int dy = 0;
    float val = 0;
    uchar dir = 0;
	uchar flag1 = 0;
	uchar flag2 = 0;
	int com = 0;
	uchar center = 0;
    uchar fmap = 0;
    // uchar &a = flag1;
    // uchar &b = flag2;
    // uchar &c = center;
    __shared__ volatile uchar sblur[32*32];
    __shared__ volatile uchar sgMap[32*32];
    // 以上 4.362 ms
    // 数据写入共享内存 7.5 ms
    if(gx<gcols && gy<grows)
        sblur LIDX(lx, ly) = blur GIDX(gx, gy);
    __syncthreads();
    // 以上 11.767 ms
    // 梯度计算 17.6 ms
	if(lx!=0 && ly!=0 && lx<(lcols-1) && ly<(lrows-1) && gx<(gcols-1) && gy<(grows-1))
    {
        dx = sblur LIDX(lx+1,ly-1);
        dx += 2 * sblur LIDX(lx+1,ly);
        dx += sblur LIDX(lx+1,ly+1);
        dx -= sblur LIDX(lx-1,ly-1);
        dx -= 2 * sblur LIDX(lx-1,ly);
        dx -= sblur LIDX(lx-1,ly+1);
        dx = abs(dx);

        dy = sblur LIDX(lx-1,ly-1);
        dy += 2 * sblur LIDX(lx,ly-1);
        dy += sblur LIDX(lx+1,ly-1);
        dy -= sblur LIDX(lx-1,ly+1);
        dy -= 2 * sblur LIDX(lx,ly+1);
        dy -= sblur LIDX(lx+1,ly+1);
        dy = abs(dy);

        val = 0.5f*dx + 0.5f*dy;
        if (val > 255) val = 255.0f;

        center = (uchar)(val);
        sgMap LIDX(lx,ly) = center;
        gMap GIDX(gx,gy) = center;

        // 1 -- vertical   0 -- horizonal
        dir = dx > dy;
        fmap |= (dir<<7)&0x80;
    }
	__syncthreads();
    // 以上 29.3 ms
    // 锚点提取 21.341 ms
	if((lx>1 || gx==1) && (ly>1 || gy==1) && (lx<(lcols-2) || gx==(gcols-2)) && (ly<(lrows-2) || gy==(grows-2)) && gx<(gcols-1) && gy<(grows-1))
	{
		// h
		flag1 = !dir;
		com = center;
		com -= sgMap LIDX(lx, ly-1);
		flag1 &= com>=ANCHOR_TH;
		com = center;
		com -= sgMap LIDX(lx, ly+1);
		flag1 &= com>=ANCHOR_TH;
		// v
		flag2 = dir;
		com = center;
		com -= sgMap LIDX(lx-1,ly);
		flag2 &= com >= ANCHOR_TH;
		com = center;
		com -= sgMap LIDX(lx+1,ly);
		flag2 &= com >= ANCHOR_TH;
        fmap |= (((flag1 | flag2) && ((gx-1)%K)==0 && ((gy-1)%K)==0)<<6)&0x40;

        // 方向分析
        // 方案1 用于GPU计算结果
        // 	0	0	0	0	0	0	0	0
        // 					----- A方向（左/上）上/左、下/右
        // 							----- B方向（右/下）上/左、下/右

        // dir : 1 -- vertical   0 -- horizonal

        //      第一轮       第二轮
        //   h-0    v-1    h-0   v-1
        // a 左上 / 上左 | 右上 / 下左
        // b 左   / 上   | 右   / 下
        // c 左下 / 上右 | 右下 / 下右
        // a = sgMap LIDX(lx-1, ly-1);
        // b = sgMap LIDX(lx-1, ly) * !dir;
        // b += sgMap LIDX(lx, ly-1) * dir;
        // c = sgMap LIDX(lx-1, ly+1) * !dir;
        // c += sgMap LIDX(lx+1, ly-1) * dir;

        // fmap |= (a>b && a>c)

        fMap GIDX(gx,gy) = fmap;
	}
    // 以上 50.641 ms

}