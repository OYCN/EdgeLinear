#include "hip/hip_runtime.h"
#include "BlockLinear.h"

__global__ void kernel(POINT *edge_set_d, int *edge_offset_d, int edge_offset_len, bool *flags_d, float epsilon);

void BlockLinear::init()
{
    HANDLE_ERROR(hipMalloc(&edge_set_d, sizeof(POINT)*rows*cols));
	HANDLE_ERROR(hipMalloc(&edge_offset_d, sizeof(int)*(rows*cols+1)));
	HANDLE_ERROR(hipMalloc(&flags_d, sizeof(bool)*rows*cols));
    if(returnH)
        HANDLE_ERROR(hipHostMalloc(&flags_h, sizeof(bool)*rows*cols));
}

void BlockLinear::deinit()
{
    HANDLE_ERROR(hipFree(edge_set_d));
	HANDLE_ERROR(hipFree(edge_offset_d));
	HANDLE_ERROR(hipFree(flags_d));
    if(returnH)
        HANDLE_ERROR(hipHostFree(flags_h));
}

void BlockLinear::enqueue(_EDoutput fMaph, cv::cuda::Stream& cvstream)
{
    const dim3 dimBlock(32,1);
    const dim3 dimGrid((cols*rows+31) / 32, 1);

    hipStream_t custream = cv::cuda::StreamAccessor::getStream(cvstream);

    HANDLE_ERROR(hipMemcpyAsync(edge_set_d, fMaph.edge_set, sizeof(POINT)*(fMaph.edge_offset)[(fMaph.edge_offset_len)-1], hipMemcpyHostToDevice, custream));
	HANDLE_ERROR(hipMemcpyAsync(edge_offset_d, fMaph.edge_offset, sizeof(int)*(fMaph.edge_offset_len), hipMemcpyHostToDevice, custream));
	HANDLE_ERROR(hipMemsetAsync(flags_d, false, sizeof(bool)*rows*cols, custream));

    kernel<<<dimGrid, dimBlock, 0, custream>>>(edge_set_d, edge_offset_d, fMaph.edge_offset_len, flags_d, th);
	// HANDLE_ERROR(hipDeviceSynchronize());
    if(returnH)
	    HANDLE_ERROR(hipMemcpyAsync(flags_h, flags_d, sizeof(bool)*(fMaph.edge_offset)[(fMaph.edge_offset_len)-1], hipMemcpyDeviceToHost, custream));


}

__global__ void kernel(POINT *edge_set_d, int *edge_offset_d, int edge_offset_len, bool *flags_d, float epsilon)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index>=(edge_offset_len-1)) return;

    float now_len = 0;
    float now_dis = 0;
    // A 为上一直线化的点，或起始点
    // B 为当前遍历的点
    // T 为上一个点
    POINT A, B, T;

    A = edge_set_d[edge_offset_d[index]];
    // 起始点置位
    flags_d[edge_offset_d[index]] = true;
    for(int j = (edge_offset_d[index] + 1); j < edge_offset_d[index + 1]; j++)
    {
        B = edge_set_d[j];
        T = edge_set_d[j-1];
        float dx = T.x - B.x;
        float dy = T.y - B.y;
        now_len += sqrt(dx * dx + dy * dy);
        dx = A.x - B.x;
        dy = A.y - B.y;
        now_dis = sqrt(dx * dx + dy * dy);
        // 若本次超过阈值，上次的为最佳点
        if(fabs(now_len - now_dis) > epsilon)
        {
            flags_d[j - 1] = true;
            // std::cout << j - 1 << ":(" << T.x << "," << T.y << ")" <<std::endl;
            // 上次点为起始点
            A = T;
            now_len = 0;
            // 需要重新计算本点
            j--;
        }
    }
    // 结束点为最佳点
    flags_d[edge_offset_d[index + 1] - 1] = true;
}
